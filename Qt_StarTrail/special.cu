#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <opencv2/opencv.hpp>
#include <opencv2/highgui/highgui.hpp>
#include "special.cuh"
#include <cmath>

using namespace cv;
using namespace std;

__global__ void addBackground(uchar* star_trail, uchar* background, uchar* output, int rows, int cols)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < cols && y < rows)
    {
        for (int c = 0; c < 3; ++c)
        {
            if (star_trail[(y * cols + x) * 3 + c] == 0)
            {
                output[(y * cols + x) * 3 + c] = background[(y * cols + x) * 3 + c];
            }
            else
            {
                output[(y * cols + x) * 3 + c] = star_trail[(y * cols + x) * 3 + c];
            }
        }
    }
}
__global__ void addForeground(uchar* star_trail, uchar* foreground, uchar* foreground_mack, uchar* output, int rows, int cols)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < cols && y < rows)
    {
        for (int c = 0; c < 3; ++c)
        {
            if (foreground_mack[(y * cols + x) * 3 + c] != 0)
            {
                output[(y * cols + x) * 3 + c] = star_trail[(y * cols + x) * 3 + c];
            }
            else
            {
                output[(y * cols + x) * 3 + c] = foreground[(y * cols + x) * 3 + c];
            }
        }
    }
}
__global__ void scaleImage(uchar* input, uchar* record, int rows, int cols, float scale, int x_center, int y_center)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate center coordinates
    int centerX = x_center;
    int centerY = y_center;

    // Calculate relative coordinates to the center
    int relX = x - centerX;
    int relY = y - centerY;

    // Apply scaling around the center
    float scale_d = 1.0 / (0.5 * scale);
    int new_x = static_cast<int>(relX * scale_d) + centerX;
    int new_y = static_cast<int>(relY * scale_d) + centerY;

    if (x < cols && y < rows && new_x >= 0 && new_x < cols && new_y >= 0 && new_y < rows)
    {
        for (int c = 0; c < 3; ++c)
        {
            // Copy pixel value to the output image
            if ((input[(new_y * cols + new_x) * 3 + c] + record[(y * cols + x) * 3 + c]) > 255)
            {
                record[(y * cols + x) * 3 + c] = 255;
            }
            else
            {
                record[(y * cols + x) * 3 + c] = input[(new_y * cols + new_x) * 3 + c] + record[(y * cols + x) * 3 + c];
            }
        }
    }
}
__global__ void rotateandscale(uchar* input, uchar* rotat, int rows, int cols, float angle, int x_center, int y_center)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < cols && y < rows)
    {
        // Calculate center coordinates
        int centerX = x_center;
        int centerY = y_center;

        // Calculate relative coordinates to the center
        int relX = x - centerX;
        int relY = y - centerY;

        // Apply scaling around the center
        float scale_d = (angle * 0.5);
        int scaledX = static_cast<int>(relX * scale_d);
        int scaledY = static_cast<int>(relY * scale_d);

        // Apply rotation around the center
        float radians = angle * 3.141592653589793;
        int rotatedX = static_cast<int>(cos(radians) * scaledX - sin(radians) * scaledY) + centerX;
        int rotatedY = static_cast<int>(sin(radians) * scaledX + cos(radians) * scaledY) + centerY;

        if (rotatedX >= 0 && rotatedX < cols && rotatedY >= 0 && rotatedY < rows)
        {
            for (int c = 0; c < 3; ++c)
            {
                if ((input[(rotatedY * cols + rotatedX) * 3 + c] + rotat[(y * cols + x) * 3 + c]) > 255)
                {
                    rotat[(y * cols + x) * 3 + c] = 255;
                }
                else
                {
                    rotat[(y * cols + x) * 3 + c] = input[(rotatedY * cols + rotatedX) * 3 + c] + rotat[(y * cols + x) * 3 + c];
                }
            }
        }
        else
        {
            for (int c = 0; c < 3; ++c)
            {
                rotat[(y * cols + x) * 3 + c] = rotat[(y * cols + x) * 3 + c];
            }
        }
    }
}

__global__ void rotateImage(uchar* input, uchar* rotat, int rows, int cols, float angle, int x_center, int y_center)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < cols && y < rows)
    {
        float radians = angle * 3.141592653589793 / 180.0;
        int new_x = static_cast<int>(cos(radians) * (x - x_center) - sin(radians) * (y - y_center) + x_center);
        int new_y = static_cast<int>(sin(radians) * (x - x_center) + cos(radians) * (y - y_center) + y_center);

        if (new_x >= 0 && new_x < cols && new_y >= 0 && new_y < rows)
        {
            for (int c = 0; c < 3; ++c)
            {
                if ((input[(new_y * cols + new_x) * 3 + c] + rotat[(y * cols + x) * 3 + c]) > 255)
                {
                    rotat[(y * cols + x) * 3 + c] = 255;
                }
                else
                {
                    rotat[(y * cols + x) * 3 + c] = input[(new_y * cols + new_x) * 3 + c] + rotat[(y * cols + x) * 3 + c];
                }
            }
        }
        else
        {
            for (int c = 0; c < 3; ++c)
            {
                rotat[(y * cols + x) * 3 + c] = rotat[(y * cols + x) * 3 + c];
            }
        }
    }
}
void Circle_rotate(Mat img1, Mat seg_img, Mat mask_img, Mat img, Mat& img_out, int x_center, int y_center)
{
    Mat foreground = img - seg_img;
	uchar* d_img1, * d_rotat, * d_img, * d_seg_img, * d_tmp, * d_mask_img, * d_foreground;
	int size = img1.cols * img1.rows * 3;
	hipMalloc((void**)&d_img1, size * sizeof(uchar));
	hipMalloc((void**)&d_rotat, size * sizeof(uchar));
    hipMalloc((void**)&d_img, size * sizeof(uchar));
    hipMalloc((void**)&d_seg_img, size * sizeof(uchar));
    hipMalloc((void**)&d_tmp, size * sizeof(uchar));
    hipMalloc((void**)&d_mask_img, size * sizeof(uchar));
    hipMalloc((void**)&d_foreground, size * sizeof(uchar));

	hipMemcpy(d_img1, img1.data, size * sizeof(uchar), hipMemcpyHostToDevice);
    hipMemcpy(d_seg_img, seg_img.data, size * sizeof(uchar), hipMemcpyHostToDevice);
    hipMemcpy(d_mask_img, mask_img.data, size * sizeof(uchar), hipMemcpyHostToDevice);
    hipMemcpy(d_foreground, foreground.data, size * sizeof(uchar), hipMemcpyHostToDevice);

	dim3 blockSize(32, 32);
	dim3 gridSize((img1.cols + blockSize.x - 1) / blockSize.x, (img1.rows + blockSize.y - 1) / blockSize.y);
	for (float angle = 0.0; angle < 240.0; angle += 0.5)
	{
        rotateImage<<<gridSize, blockSize>>>(d_img1, d_rotat, img1.rows, img1.cols, angle, x_center, y_center);
	}
    addBackground<<<gridSize, blockSize>>>(d_rotat, d_seg_img, d_tmp, img1.rows, img1.cols);
    addForeground<<<gridSize, blockSize>>>(d_tmp, d_foreground, d_mask_img, d_rotat, img1.rows, img1.cols);
	uchar* rotated_image = new uchar[size];
	hipMemcpy(rotated_image, d_rotat, size * sizeof(uchar), hipMemcpyDeviceToHost);
	Mat rotated_mat(img1.rows, img1.cols, CV_8UC3, rotated_image);
    //imshow("", rotated_mat);
    //waitKey(0);
	img_out = rotated_mat.clone();

    hipFree(d_img);
    hipFree(d_img1);
    hipFree(d_rotat);
    hipFree(d_seg_img);
    hipFree(d_tmp);
    hipFree(d_mask_img);
    hipFree(d_foreground);
}
void Radiation_rotate(Mat img1, Mat seg_img, Mat mask_img, Mat img, Mat& img_out, int x_center, int y_center)
{
    Mat foreground = img - seg_img;
    uchar* d_img1, * d_rotat, * d_img, * d_seg_img, * d_tmp, * d_mask_img, * d_foreground;
    int size = img1.cols * img1.rows * 3;
    hipMalloc((void**)&d_img1, size * sizeof(uchar));
    hipMalloc((void**)&d_rotat, size * sizeof(uchar));
    hipMalloc((void**)&d_img, size * sizeof(uchar));
    hipMalloc((void**)&d_seg_img, size * sizeof(uchar));
    hipMalloc((void**)&d_tmp, size * sizeof(uchar));
    hipMalloc((void**)&d_mask_img, size * sizeof(uchar));
    hipMalloc((void**)&d_foreground, size * sizeof(uchar));

    hipMemcpy(d_img1, img1.data, size * sizeof(uchar), hipMemcpyHostToDevice);
    hipMemcpy(d_seg_img, seg_img.data, size * sizeof(uchar), hipMemcpyHostToDevice);
    hipMemcpy(d_mask_img, mask_img.data, size * sizeof(uchar), hipMemcpyHostToDevice);
    hipMemcpy(d_foreground, foreground.data, size * sizeof(uchar), hipMemcpyHostToDevice);

    dim3 blockSize(32, 32);
    dim3 gridSize((img1.cols + blockSize.x - 1) / blockSize.x, (img1.rows + blockSize.y - 1) / blockSize.y);
    for (float angle = 2; angle <= 4; angle += 0.01)
    {
        scaleImage << <gridSize, blockSize >> > (d_img1, d_rotat, img1.rows, img1.cols, angle, x_center, y_center);
    }
    addBackground << <gridSize, blockSize >> > (d_rotat, d_seg_img, d_tmp, img1.rows, img1.cols);
    addForeground << <gridSize, blockSize >> > (d_tmp, d_foreground, d_mask_img, d_rotat, img1.rows, img1.cols);
    uchar* rotated_image = new uchar[size];
    hipMemcpy(rotated_image, d_rotat, size * sizeof(uchar), hipMemcpyDeviceToHost);
    Mat rotated_mat(img1.rows, img1.cols, CV_8UC3, rotated_image);
    //imshow("", rotated_mat);
    //waitKey(0);
    img_out = rotated_mat.clone();

    hipFree(d_img);
    hipFree(d_img1);
    hipFree(d_rotat);
    hipFree(d_seg_img);
    hipFree(d_tmp);
    hipFree(d_mask_img);
    hipFree(d_foreground);
}
void Spiral_rotate(Mat img1, Mat seg_img, Mat mask_img, Mat img, Mat& img_out, int x_center, int y_center)
{
    Mat foreground = img - seg_img;
    uchar* d_img1, * d_rotat, * d_img, * d_seg_img, * d_tmp, * d_mask_img, * d_foreground;
    int size = img1.cols * img1.rows * 3;
    hipMalloc((void**)&d_img1, size * sizeof(uchar));
    hipMalloc((void**)&d_rotat, size * sizeof(uchar));
    hipMalloc((void**)&d_img, size * sizeof(uchar));
    hipMalloc((void**)&d_seg_img, size * sizeof(uchar));
    hipMalloc((void**)&d_tmp, size * sizeof(uchar));
    hipMalloc((void**)&d_mask_img, size * sizeof(uchar));
    hipMalloc((void**)&d_foreground, size * sizeof(uchar));

    hipMemcpy(d_img1, img1.data, size * sizeof(uchar), hipMemcpyHostToDevice);
    hipMemcpy(d_seg_img, seg_img.data, size * sizeof(uchar), hipMemcpyHostToDevice);
    hipMemcpy(d_mask_img, mask_img.data, size * sizeof(uchar), hipMemcpyHostToDevice);
    hipMemcpy(d_foreground, foreground.data, size * sizeof(uchar), hipMemcpyHostToDevice);

    dim3 blockSize(32, 32);
    dim3 gridSize((img1.cols + blockSize.x - 1) / blockSize.x, (img1.rows + blockSize.y - 1) / blockSize.y);
    for (float angle = 2; angle <= 4; angle += 0.002)
    {
        rotateandscale << <gridSize, blockSize >> > (d_img1, d_rotat, img1.rows, img1.cols, angle, x_center, y_center);
    }
    addBackground << <gridSize, blockSize >> > (d_rotat, d_seg_img, d_tmp, img1.rows, img1.cols);
    addForeground << <gridSize, blockSize >> > (d_tmp, d_foreground, d_mask_img, d_rotat, img1.rows, img1.cols);
    uchar* rotated_image = new uchar[size];
    hipMemcpy(rotated_image, d_rotat, size * sizeof(uchar), hipMemcpyDeviceToHost);
    Mat rotated_mat(img1.rows, img1.cols, CV_8UC3, rotated_image);
    //imshow("", rotated_mat);
    //waitKey(0);
    img_out = rotated_mat.clone();

    hipFree(d_img);
    hipFree(d_img1);
    hipFree(d_rotat);
    hipFree(d_seg_img);
    hipFree(d_tmp);
    hipFree(d_mask_img);
    hipFree(d_foreground);
}
void Circular_GIF(Mat img1, Mat seg_img, Mat mask_img, Mat img, int x_center, int y_center, string outpath, string basename)
{
    cv::VideoWriter writer;
    string path = "tmp.avi";
    writer.open(path.c_str(), VideoWriter::fourcc('M', 'P', '4', '2'), 30, Size(img1.cols, img1.rows));

    Mat foreground = img - seg_img;
    uchar* d_img1, * d_rotat, * d_img, * d_seg_img, * d_tmp, * d_mask_img, * d_foreground;
    int size = img1.cols * img1.rows * 3;
    hipMalloc((void**)&d_img1, size * sizeof(uchar));
    hipMalloc((void**)&d_rotat, size * sizeof(uchar));
    hipMalloc((void**)&d_img, size * sizeof(uchar));
    hipMalloc((void**)&d_seg_img, size * sizeof(uchar));
    hipMalloc((void**)&d_tmp, size * sizeof(uchar));
    hipMalloc((void**)&d_mask_img, size * sizeof(uchar));
    hipMalloc((void**)&d_foreground, size * sizeof(uchar));

    hipMemcpy(d_img1, img1.data, size * sizeof(uchar), hipMemcpyHostToDevice);
    hipMemcpy(d_seg_img, seg_img.data, size * sizeof(uchar), hipMemcpyHostToDevice);
    hipMemcpy(d_mask_img, mask_img.data, size * sizeof(uchar), hipMemcpyHostToDevice);
    hipMemcpy(d_foreground, foreground.data, size * sizeof(uchar), hipMemcpyHostToDevice);

    dim3 blockSize(32, 32);
    dim3 gridSize((img1.cols + blockSize.x - 1) / blockSize.x, (img1.rows + blockSize.y - 1) / blockSize.y);
    for (float angle = 0.0; angle < 240.0; angle += 0.5)
    {
        rotateImage << <gridSize, blockSize >> > (d_img1, d_rotat, img1.rows, img1.cols, angle, x_center, y_center);
        addBackground << <gridSize, blockSize >> > (d_rotat, d_seg_img, d_tmp, img1.rows, img1.cols);
        addForeground << <gridSize, blockSize >> > (d_tmp, d_foreground, d_mask_img, d_rotat, img1.rows, img1.cols);

        uchar* rotated_image = new uchar[size];
        hipMemcpy(rotated_image, d_rotat, size * sizeof(uchar), hipMemcpyDeviceToHost);
        Mat rotated_mat(img1.rows, img1.cols, CV_8UC3, rotated_image);
        writer.write(rotated_mat);

        delete[] rotated_image;
    }
    writer.release();
    string ffmpegCommand = "E:\\ffmpeg\\bin\\ffmpeg.exe -i " + std::string("tmp.avi") + " " + outpath + "/Circular_" + basename + ".gif";
    system(ffmpegCommand.c_str());
    remove("tmp.avi");

    hipFree(d_img);
    hipFree(d_img1);
    hipFree(d_rotat);
    hipFree(d_seg_img);
    hipFree(d_tmp);
    hipFree(d_mask_img);
    hipFree(d_foreground);
    /*
    delete[] d_img1;
    delete[] d_rotat;
    delete[] d_img;
    delete[] d_seg_img;
    delete[] d_tmp;
    delete[] d_mask_img;
    delete[] d_foreground;
    */
}
void Radiation_GIF(Mat img1, Mat seg_img, Mat mask_img, Mat img, int x_center, int y_center, string outpath, string basename)
{
    cv::VideoWriter writer;
    string path = "tmp.avi";
    writer.open(path.c_str(), VideoWriter::fourcc('M', 'P', '4', '2'), 30, Size(img1.cols, img1.rows));

    Mat foreground = img - seg_img;
    uchar* d_img1, * d_rotat, * d_img, * d_seg_img, * d_tmp, * d_mask_img, * d_foreground;
    int size = img1.cols * img1.rows * 3;
    hipMalloc((void**)&d_img1, size * sizeof(uchar));
    hipMalloc((void**)&d_rotat, size * sizeof(uchar));
    hipMalloc((void**)&d_img, size * sizeof(uchar));
    hipMalloc((void**)&d_seg_img, size * sizeof(uchar));
    hipMalloc((void**)&d_tmp, size * sizeof(uchar));
    hipMalloc((void**)&d_mask_img, size * sizeof(uchar));
    hipMalloc((void**)&d_foreground, size * sizeof(uchar));

    hipMemcpy(d_img1, img1.data, size * sizeof(uchar), hipMemcpyHostToDevice);
    hipMemcpy(d_seg_img, seg_img.data, size * sizeof(uchar), hipMemcpyHostToDevice);
    hipMemcpy(d_mask_img, mask_img.data, size * sizeof(uchar), hipMemcpyHostToDevice);
    hipMemcpy(d_foreground, foreground.data, size * sizeof(uchar), hipMemcpyHostToDevice);

    dim3 blockSize(32, 32);
    dim3 gridSize((img1.cols + blockSize.x - 1) / blockSize.x, (img1.rows + blockSize.y - 1) / blockSize.y);
    for (float angle = 2; angle <= 4; angle += 0.01)
    {
        scaleImage << <gridSize, blockSize >> > (d_img1, d_rotat, img1.rows, img1.cols, angle, x_center, y_center);
        addBackground << <gridSize, blockSize >> > (d_rotat, d_seg_img, d_tmp, img1.rows, img1.cols);
        addForeground << <gridSize, blockSize >> > (d_tmp, d_foreground, d_mask_img, d_rotat, img1.rows, img1.cols);

        uchar* rotated_image = new uchar[size];
        hipMemcpy(rotated_image, d_rotat, size * sizeof(uchar), hipMemcpyDeviceToHost);
        Mat rotated_mat(img1.rows, img1.cols, CV_8UC3, rotated_image);
        writer.write(rotated_mat);
        delete[] rotated_image;
    }
    writer.release();
    string ffmpegCommand = "E:\\ffmpeg\\bin\\ffmpeg.exe -i " + std::string("tmp.avi") + " " + outpath + "/Radiation_" + basename + ".gif";
    system(ffmpegCommand.c_str());
    remove("tmp.avi");

    hipFree(d_img);
    hipFree(d_img1);
    hipFree(d_rotat);
    hipFree(d_seg_img);
    hipFree(d_tmp);
    hipFree(d_mask_img);
    hipFree(d_foreground);
}
void Spiral_GIF(Mat img1, Mat seg_img, Mat mask_img, Mat img, int x_center, int y_center, string outpath, string basename)
{
    cv::VideoWriter writer;
    string path = "tmp.avi";
    writer.open(path.c_str(), VideoWriter::fourcc('M', 'P', '4', '2'), 30, Size(img1.cols, img1.rows));

    Mat foreground = img - seg_img;
    uchar* d_img1, * d_rotat, * d_img, * d_seg_img, * d_tmp, * d_mask_img, * d_foreground;
    int size = img1.cols * img1.rows * 3;
    hipMalloc((void**)&d_img1, size * sizeof(uchar));
    hipMalloc((void**)&d_rotat, size * sizeof(uchar));
    hipMalloc((void**)&d_img, size * sizeof(uchar));
    hipMalloc((void**)&d_seg_img, size * sizeof(uchar));
    hipMalloc((void**)&d_tmp, size * sizeof(uchar));
    hipMalloc((void**)&d_mask_img, size * sizeof(uchar));
    hipMalloc((void**)&d_foreground, size * sizeof(uchar));

    hipMemcpy(d_img1, img1.data, size * sizeof(uchar), hipMemcpyHostToDevice);
    hipMemcpy(d_seg_img, seg_img.data, size * sizeof(uchar), hipMemcpyHostToDevice);
    hipMemcpy(d_mask_img, mask_img.data, size * sizeof(uchar), hipMemcpyHostToDevice);
    hipMemcpy(d_foreground, foreground.data, size * sizeof(uchar), hipMemcpyHostToDevice);

    dim3 blockSize(32, 32);
    dim3 gridSize((img1.cols + blockSize.x - 1) / blockSize.x, (img1.rows + blockSize.y - 1) / blockSize.y);
    for (float angle = 2; angle <= 4; angle += 0.002)
    {
        //scaleImage << <gridSize, blockSize >> > (d_img1, d_rotat, img1.rows, img1.cols, angle, x_center, y_center);
        //rotateImage << <gridSize, blockSize >> > (d_rotat, d_rotat, img1.rows, img1.cols, angle, x_center, y_center);
        rotateandscale << <gridSize, blockSize >> > (d_img1, d_rotat, img1.rows, img1.cols, angle, x_center, y_center);
        addBackground << <gridSize, blockSize >> > (d_rotat, d_seg_img, d_tmp, img1.rows, img1.cols);
        addForeground << <gridSize, blockSize >> > (d_tmp, d_foreground, d_mask_img, d_rotat, img1.rows, img1.cols);

        uchar* rotated_image = new uchar[size];
        hipMemcpy(rotated_image, d_rotat, size * sizeof(uchar), hipMemcpyDeviceToHost);
        Mat rotated_mat(img1.rows, img1.cols, CV_8UC3, rotated_image);
        writer.write(rotated_mat);
        delete[] rotated_image;
    }
    writer.release();
    string ffmpegCommand = "E:\\ffmpeg\\bin\\ffmpeg.exe -i " + std::string("tmp.avi") + " " + outpath + "/Spiral_" + basename + ".gif";
    system(ffmpegCommand.c_str());
    remove("tmp.avi");

    hipFree(d_img);
    hipFree(d_img1);
    hipFree(d_rotat);
    hipFree(d_seg_img);
    hipFree(d_tmp);
    hipFree(d_mask_img);
    hipFree(d_foreground);
}